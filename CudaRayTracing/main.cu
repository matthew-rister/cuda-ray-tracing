#include "hip/hip_runtime.h"
﻿#include <cstdint>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <memory>
#include <string_view>

#include <hip/hip_runtime.h>
#include <>
#include <fmt/core.h>
#include <glm/glm.hpp>

#include "rt/camera.h"
#include "rt/image.h"
#include "rt/ray.h"
#include "rt/sphere.h"

using namespace fmt;
using namespace glm;
using namespace rt;
using namespace std;

#define __FILENAME__ (strrchr(__FILE__, '\\') ? strrchr(__FILE__, '\\') + 1 : __FILE__)
#define CHECK_CUDA_ERRORS(status) CheckCudaErrors((status), #status, __FILENAME__, __LINE__)

void CheckCudaErrors(
	const hipError_t status, const std::string_view function, const std::string_view filename, const int line_number) {

	if (status != hipSuccess) {
		throw runtime_error{
			format("{} failed at {}:{} with error \"{}\"", function, filename, line_number, hipGetErrorString(status))
		};
	}
}

__device__ vec3 ComputeRayColor(const Ray& ray, const Sphere& sphere) noexcept {
	if (sphere.Intersect(ray)) return vec3{1.f, 0.f, 0.f};
	const auto t = .5f * (ray.direction().y + 1.f);
	return (1.f - t) * vec3{1.f} + t * vec3{.5f, .7f, 1.f};
}

__global__ void Render(const Image image, const Camera camera, const Sphere sphere, uint8_t* const frame_buffer) {
	const auto [width, height, channels, max_color_value] = image;
	const auto i = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);
	const auto j = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
	const auto k = i * width * channels + j * channels;

	if (i < height && j < width) {
		const auto u = static_cast<float>(j) / static_cast<float>(width - 1);
		const auto v = static_cast<float>(i) / static_cast<float>(height - 1);
		const auto ray = camera.RayThrough(u, v);
		const auto color = static_cast<float>(max_color_value) * ComputeRayColor(ray, sphere);
		frame_buffer[k] = static_cast<uint8_t>(color.r);
		frame_buffer[k + 1] = static_cast<uint8_t>(color.g);
		frame_buffer[k + 2] = static_cast<uint8_t>( color.b);
	}
}

int main() {

	try {
		constexpr auto kAspectRatio = 16.f / 9.f;
		const Camera camera{vec3{0.f}, kAspectRatio};

		constexpr auto kImageHeight = 400;
		constexpr auto kImageWidth = static_cast<int>(kAspectRatio * kImageHeight);
		constexpr auto kColorChannels = 3;
		constexpr auto kImageSize = static_cast<int64_t>(kImageWidth) * kImageHeight * kColorChannels;
		constexpr auto kImageSizeBytes = kImageSize * sizeof(uint8_t);
		constexpr auto kMaxColorValue = numeric_limits<uint8_t>::max();
		const Image image{kImageWidth, kImageHeight, kColorChannels, kMaxColorValue};
		uint8_t* device_frame_buffer = nullptr;
		CHECK_CUDA_ERRORS(hipMallocManaged(reinterpret_cast<void**>(&device_frame_buffer), kImageSizeBytes));

		const glm::vec3 origin{0.f, 0.f, -1.f};
		const Sphere sphere{origin, .5f};

		const dim3 threads{16, 16};
		const dim3 blocks{kImageWidth / threads.x + 1, kImageHeight / threads.y + 1};
		Render<<<blocks, threads>>>(image, camera, sphere, device_frame_buffer);

		CHECK_CUDA_ERRORS(hipGetLastError());
		CHECK_CUDA_ERRORS(hipDeviceSynchronize());

		const unique_ptr<uint8_t[]> host_frame_buffer{new uint8_t[kImageSize]};
		CHECK_CUDA_ERRORS(hipMemcpy(host_frame_buffer.get(), device_frame_buffer, kImageSizeBytes, hipMemcpyDefault));
		CHECK_CUDA_ERRORS(hipFree(device_frame_buffer));

		image.SaveAs(host_frame_buffer.get(), "img/ch5.png");
		return EXIT_SUCCESS;

	} catch (exception& e) {
		cerr << e.what();
		hipDeviceReset();
		return EXIT_FAILURE;
	}
}
